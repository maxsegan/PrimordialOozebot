#include "hip/hip_runtime.h"
#include <math.h>
#include <vector>
#include <map>
#include <time.h>
#include "OozebotEncoding.h"
#include "ParetoSelector.h"
//#include <chrono>

// Usage: nvcc -O2 evoAlgo.cu -o evoAlgo -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.27.29110\bin\Hostx64\x64" cudaSim.cu OozebotEncoding.cpp ParetoSelector.cpp ParetoFront.cpp

// TODO command line args
// TODO air/water resistence

int main() {
    // TODO objectives - fitness, age (in log tenure groupings maybe?), weight?
    // Meta objectives to consider
    // – Simplicity
    // – Evolvability
    // – Novelty / Diversity
    // – Robustness / sensitivity

    srand(time(NULL));

    int maxEvaluations = 100000; // TODO take as a param
    const int minNumSolutions = 300; // TODO take as a param
    double mutationRate = 0.05; // TODO take as a param

    ParetoSelector generation(minNumSolutions, mutationRate);

    OozebotEncoding previousEncoding;
    AsyncSimHandle previousHandle;

    for (int i = 0; i <= minNumSolutions; i++) {
        OozebotEncoding encoding = previousEncoding;
        AsyncSimHandle handle = previousHandle;
        if (i != minNumSolutions) {
            previousEncoding = OozebotEncoding::randomEncoding();
            previousHandle = OozebotEncoding::evaluate(previousEncoding, i);
        }
        if (i > 0) {
            printf("Evaluating %d\n", i - 1);
            auto res = OozebotEncoding::wait(handle);
            encoding.fitness = res.first;
            printf("Fitness was %f\n", encoding.fitness);
            encoding.numTouchesRatio = res.second;
            generation.globalParetoFront.evaluateEncoding(encoding);
            generation.insertOozebot(encoding);
        }
    }

    int numEvaluations = minNumSolutions;
    // In this stage do baseball leagues too, maybe 100k iterations, then create another one (recursive) as it's competitor
    while (numEvaluations < maxEvaluations) {
        numEvaluations += generation.selectAndMate();
        printf("Finished run #%d\n", numEvaluations);
    }
    // TODO hill climb at the end of each generation
    return 0;
}
