#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <map>
#include <chrono>
#include <hip/hip_runtime.h>

#include "cudaSim.h"

// Usage: nvcc -O2 cudaSim.cu -o cudaSim -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.27.29110\bin\Hostx64\x64"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define staticFriction 0.5
#define kineticFriction 0.3
#define dt 0.0001
#define dampening 0.9995
#define gravity -9.81
#define kGround -100000.0

__global__ void update_spring(
    Point *points,
    Spring *springs,
    SpringDelta *springDeltas,
    int n,
    double preset0,
    double preset1,
    double preset2,
    double preset3,
    double preset4,
    double preset5) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    Spring s = springs[i];
    Point p1 = points[s.p1];
    Point p2 = points[s.p2];

    float dx = p1.x - p2.x;
    float dy = p1.y - p2.y;
    float dz = p1.z - p2.z;

    float dist = sqrt(dx * dx + dy * dy + dz * dz);

    // negative if repelling, positive if attracting
    float adjust;
    switch (s.flexIndex) {
        case 0:
            adjust = preset0;
            break;
        case 1:
            adjust = preset1;
            break;
        case 2:
            adjust = preset2;
            break;
        case 3:
            adjust = preset3;
            break;
        case 4:
            adjust = preset4;
            break;
        case 5:
            adjust = preset5;
            break;
        default:
            adjust = 1;
            break;
    }
    float f = s.k * (dist - (s.l0 * adjust));

    float fd = f / dist;

    float xd = fd * dx;
    float yd = fd * dy;
    float zd = fd * dz;

    springDeltas[p1.springDeltaIndex + s.p1SpringIndex] = {-xd, -yd, -zd};
    springDeltas[p2.springDeltaIndex + s.p2SpringIndex] = {xd, yd, zd};
}

__global__ void update_point(Point *points, SpringDelta *springDeltas, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    Point p = points[i];

	float mass = p.mass;
    float fx = 0;
    float fz = 0;
    float fy = gravity * mass;
    int startIndex = p.springDeltaIndex;
    int done = p.numSprings + startIndex;
    for (int j = startIndex; j < done; j++) {
        SpringDelta sd = springDeltas[j];

		fx += sd.dx;
    	fy += sd.dy;
    	fz += sd.dz;
	}

    float y = p.y;
    float vx = p.vx;
    float vy = p.vy;
    float vz = p.vz;

    if (y <= 0) {
        float fh = sqrt(fx * fx + fz * fz);
        float fyfric = abs(fy * staticFriction);
        if (fh < fyfric) {
            fx = 0;
            fz = 0;
        } else {
            float fykinetic = abs(fy * kineticFriction) / fh;
            fx = fx - fx * fykinetic;
            fz = fz - fz * fykinetic;
        }
        fy += kGround * y;
    }
    float ax = fx / mass;
    float ay = fy / mass;
    float az = fz / mass;

    vx = (ax * dt + vx) * dampening;
    p.vx = vx;
    vy = (ay * dt + vy) * dampening;
    p.vy = vy;
    vz = (az * dt + vz) * dampening;
    p.vz = vz;
    p.x += vx * dt;
    p.y += vy * dt;
    p.z += vz * dt;
    points[i] = p;
}

AsyncSimHandle simulate(std::vector<Point> &points, std::vector<Spring> &springs, std::vector<FlexPreset> &presets, double n, double oscillationFrequency, int streamNum, double length) {
    if (points.size() == 0) {
        printf("No points, early ejecting from sim\n");
        return { {}, NULL, NULL, NULL};
    }
    std::vector<SpringDelta> pointSprings(springs.size() * 2, {0,0,0});
    int springDeltaIndex  = 0;
    for (int i = 0; i < points.size(); i++) {
        points[i].springDeltaIndex = springDeltaIndex;
        springDeltaIndex += points[i].numSprings;
    }

    int nDevices;
    int deviceNumber = 0;
    HANDLE_ERROR(hipGetDeviceCount(&nDevices));
    if (nDevices > 1) {
        deviceNumber = 1;//streamNum % nDevices;
        HANDLE_ERROR(hipSetDevice(deviceNumber));
    }

    Point *p_d;
    Spring *s_d;
    SpringDelta *ps_d;
    HANDLE_ERROR(hipMalloc(&p_d, points.size() * sizeof(Point)));
    HANDLE_ERROR(hipMemcpy(p_d, &points[0], points.size() * sizeof(Point), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc(&s_d, springs.size() * sizeof(Spring)));
    HANDLE_ERROR(hipMemcpy(s_d, &springs[0], springs.size() * sizeof(Spring), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc(&ps_d, pointSprings.size() * sizeof(SpringDelta)));
    HANDLE_ERROR(hipMemcpy(ps_d, &pointSprings[0], pointSprings.size() * sizeof(SpringDelta), hipMemcpyHostToDevice));

    double t = 0;
    int numPoints = points.size();
    int numPointThreads = 12;
    int numPointBlocks = numPoints / numPointThreads + 1;
  
    int numSprings = springs.size();
    int numSpringThreads = 25;
    int numSpringBlocks = numSprings / numSpringThreads + 1;

    std::vector<float> pv;
    for (auto it = presets.begin(); it != presets.end(); it++) {
        pv.push_back(0.0);
    }

    while (t < n) {
        for (int i = 0; i < pv.size(); i++) {
            const float a = presets[i].a;
            const float b = presets[i].b;
            const float c = presets[i].c; 
            pv[i] = a + b * sin(t * oscillationFrequency);
        }
        update_spring<<<numSpringBlocks, numSpringThreads>>>(p_d, s_d, ps_d, numSprings, pv[0], pv[1], pv[2], pv[3], pv[4], pv[5]);
        update_point<<<numPointBlocks, numPointThreads>>>(p_d, ps_d, numPoints);
        if (t < 1.0 && t + dt >= 1.0) {
            HANDLE_ERROR(hipMemcpyAsync(&points[0], p_d, numPoints * sizeof(Point), hipMemcpyDeviceToHost));
            int numCycles = 1;
            double oscillationDuration = 2 * M_PI / oscillationFrequency;
            while ((oscillationDuration * numCycles + t) < n) {
                numCycles += 1;
            }
            n = (oscillationDuration * numCycles) + t;
        }
        t += dt;
    }

    return {points, p_d, s_d, ps_d, numSprings, length, t - 1.0, deviceNumber};
}

void synchronize(AsyncSimHandle &handle) {
    HANDLE_ERROR(hipSetDevice(handle.device));
    hipDeviceSynchronize();
}

void resolveAndKeepAlive(AsyncSimHandle &handle) {
    HANDLE_ERROR(hipSetDevice(handle.device));
    HANDLE_ERROR(hipMemcpy(&handle.points[0], handle.p_d, handle.points.size() * sizeof(Point), hipMemcpyDeviceToHost));
}

void simulateAgain(AsyncSimHandle &handle, std::vector<FlexPreset> &presets, double t, double n, double oscillationFrequency, int streamNum) {
    int numPoints = handle.points.size();
    int numPointThreads = 12;
    int numPointBlocks = numPoints / numPointThreads + 1;
  
    int numSpringThreads = 25;
    int numSpringBlocks = handle.numSprings / numSpringThreads + 1;

    std::vector<float> pv;
    for (auto it = presets.begin(); it != presets.end(); it++) {
        pv.push_back(0.0);
    }

    HANDLE_ERROR(hipSetDevice(handle.device));
    while (t < n) {
        for (int i = 0; i < pv.size(); i++) {
            const float a = presets[i].a;
            const float b = presets[i].b;
            const float c = presets[i].c; 
            pv[i] = a + b * sin(t * oscillationFrequency);
        }
        update_spring<<<numSpringBlocks, numSpringThreads>>>(handle.p_d, handle.s_d, handle.ps_d, handle.numSprings, pv[0], pv[1], pv[2], pv[3], pv[4], pv[5]);
        update_point<<<numPointBlocks, numPointThreads>>>(handle.p_d, handle.ps_d, numPoints);
        t += dt;
    }
}

void resolveSim(AsyncSimHandle &handle) {
    if (handle.points.size() == 0) {
        return;
    }
    HANDLE_ERROR(hipSetDevice(handle.device));
    HANDLE_ERROR(hipMemcpy(&handle.points[0], handle.p_d, handle.points.size() * sizeof(Point), hipMemcpyDeviceToHost));
    
    HANDLE_ERROR(hipFree(handle.p_d));
    HANDLE_ERROR(hipFree(handle.s_d));
    HANDLE_ERROR(hipFree(handle.ps_d));
}
