#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <map>
#include <chrono>

// Usage: nvcc -O2 cudaSim.cu -o cudaSim -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.27.29110\bin\Hostx64\x64"

struct Point {
  double x; // meters
  double y; // meters
  double z; // meters
  double vx; // meters/second
  double vy; // meters/second
  double vz; // meters/second
  double mass; // kg
  int numSprings; // Int - hack for CUDA ease
};

struct Spring {
  double k; // N/m
  int p1; // Index of first point
  int p2; // Index of second point
  double l0; // meters
  double dx; // caching for CUDA ease
  double dy; // caching for CUDA ease
  double dz; // caching for CUDA ease
};

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<int> &pointSprings);

#define maxSprings 28
#define staticFriction 0.5
#define kineticFriction 0.3
#define dt 0.0001
#define dampening 1 - (0.0005)
#define gravity -9.81
#define kSpring 500.0
#define kGround 100000.0
const double kOscillationFrequency = 0;
const double kDropHeight = 0.2;
const int pointsPerSide = 45;

__global__ void update_spring(Point *points, Spring *springs, double adjust, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
    	return;
    }

    Point p1 = points[springs[i].p1];
    Point p2 = points[springs[i].p2];

    double p1x = p1.x;
    double p1y = p1.y;
    double p1z = p1.z;
    double p2x = p2.x;
    double p2y = p2.y;
    double p2z = p2.z;
    double dist = sqrt(pow(p1x - p2x, 2) + pow(p1y - p2y, 2) + pow(p1z - p2z, 2));

    // negative if repelling, positive if attracting
    double f = springs[i].k * (dist - (springs[i].l0 * adjust));
    // distribute force across the axes
    double dx = f * (p1x - p2x) / dist;
    double dy = f * (p1y - p2y) / dist;
    double dz = f * (p1z - p2z) / dist;

    springs[i].dx = dx;
    springs[i].dy = dy;
    springs[i].dz = dz;
}

__global__ void update_point(Point *points, Spring *springs, int *pointsToSprings) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    Point p = points[i];
    double mass = p.mass;
    double y = p.y;
    double vx = p.vx;
    double vy = p.vy;
    double vz = p.vz;
    double fy = gravity * mass;
    double fx = 0;
    double fz = 0;
    for (int j = 0; j < p.numSprings; j++) {
    	int springIndex = pointsToSprings[i * maxSprings + j];
        Spring s = springs[springIndex];
        if (s.p1 == i) {
            fx -= s.dx;
            fy -= s.dy;
            fz -= s.dz;
        } else {
            fx += s.dx;
            fy += s.dy;
            fz += s.dz;
        }
    }

    if (y <= 0) {
        double fh = sqrt(pow(fx, 2) + pow(fz, 2));
        double fyfric = abs(fy * staticFriction);
        if (fh < fyfric) {
            fx = 0;
            fz = 0;
        } else {
            double fykinetic = abs(fy * kineticFriction);
            fx = fx - fx / fh * fykinetic;
            fz = fz - fz / fh * fykinetic;
        }
        fy += -kGround * y;
    }
    double ax = fx / mass;
    double ay = fy / mass;
    double az = fz / mass;
    // reset the force cache
    vx = (ax * dt + vx) * dampening;
    p.vx = vx;
    vy = (ay * dt + vy) * dampening;
    p.vy = vy;
    vz = (az * dt + vz) * dampening;
    p.vz = vz;
    p.x += vx * dt;
    p.y += vy * dt;
    p.z += vz * dt;
    points[i] = p;
}

int main() {
    std::vector<Point> points;
    std::vector<Spring> springs;
    std::vector<int> pointSprings(pointsPerSide * pointsPerSide * pointsPerSide * maxSprings, 0);

    genPointsAndSprings(points, springs, pointSprings);

    Point *p_d;
    Spring *s_d;
    int *ps_d;
    hipMalloc(&p_d, points.size() * sizeof(Point));
    hipMemcpy(p_d, &points[0], points.size() * sizeof(Point), hipMemcpyHostToDevice);

    hipMalloc(&s_d, springs.size() * sizeof(Spring));
    hipMemcpy(s_d, &springs[0], springs.size() * sizeof(Spring), hipMemcpyHostToDevice);

    hipMalloc(&ps_d, pointSprings.size() * sizeof(int));
    hipMemcpy(ps_d, &pointSprings[0], pointSprings.size() * sizeof(int),  hipMemcpyHostToDevice);

    double t = 0;
    // 60 fps - 0.000166
    double limit = 1;
    int ppsSquare = pointsPerSide * pointsPerSide;
  
  	int numSprings = (int)springs.size();

    int springThreads = 100;
    int springBlocks = (int)ceil(numSprings / 100.0);
    printf("num springs evaluated: %lld\n", long long int(limit / dt * numSprings));
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    while (t < limit) {
        double adjust = 1 + sin(t * kOscillationFrequency) * 0.1;
        
        update_spring<<<springBlocks, springThreads>>>(p_d, s_d, adjust, numSprings);
        update_point<<<ppsSquare, pointsPerSide>>>(p_d, s_d, ps_d);
        t += dt;
    }

    hipDeviceSynchronize();
 

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
    std::cout << "Time difference = " << ms.count() / 1000.0 << "[s]" << std::endl;

    Point *ps = (Point *)malloc(points.size() * sizeof(Point));
    hipMemcpy(ps, p_d, points.size() * sizeof(Point), hipMemcpyDeviceToHost);
    for (int i = 0; i < points.size(); i++) {
    	printf("x: %f, y: %f, z: %f, %d\n", ps[i].x, ps[i].y, ps[i].z, i);
    	break;
    }
    
    hipFree(p_d);
    hipFree(s_d);
    hipFree(ps_d);
    free(ps);

    return 0;
}

void genPointsAndSprings(
	std::vector<Point> &points,
	std::vector<Spring> &springs,
	std::vector<int> &pointSprings) {
    std::map<int, std::map<int, std::map<int, Point>>> cache;

    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                // (0,0,0) or (0.1,0.1,0.1) and all combinations
                Point p = {x / 10.0, kDropHeight + y / 10.0, z / 10.0, 0, 0, 0, 0.1, 0};
                points.push_back(p);
                if (cache.count(x) == 0) {
                    cache[x] = {};
                }
                if (cache[x].count(y) == 0) {
                    cache[x][y] = {};
                }
                cache[x][y][z] = p;
            }
        }
    }
    std::map<int, std::vector<int>> connected;
    double ppsSquare = pointsPerSide * pointsPerSide;
    connected[0] = {};
    // Create the springs
    for (int x = 0; x < pointsPerSide; x++) {
        for (int y = 0; y < pointsPerSide; y++) {
            for (int z = 0; z < pointsPerSide; z++) {
                int p1index = z + pointsPerSide * y + ppsSquare * x;

                Point p1 = cache[x][y][z];
                for (int x1 = x - 1; x1 < x + 2; x1++) {
                    if (x1 == pointsPerSide || x1 < 0) {
                        continue;
                    }
                    for (int y1 = y - 1; y1 < y + 2; y1++) {
                        if (y1 == pointsPerSide || y1 < 0) {
                            continue;
                        }
                        for (int z1 = z - 1; z1 < z + 2; z1++) {
                            if (z1 == pointsPerSide || z1 < 0 || (x1 == x && y1 == y && z1 == z)) {
                                continue;
                            }
                            int p2index = z1 + pointsPerSide * y1 + ppsSquare * x1;
                            if (connected.find(p2index) == connected.end()) {
                                connected[p2index] = {};
                            }
                            if (std::find(connected[p1index].begin(), connected[p1index].end(), p2index) != connected[p1index].end()) {
                                continue;
                            }
                            connected[p1index].push_back(p2index);
                            connected[p2index].push_back(p1index);

                            Point p2 = cache[x1][y1][z1];
                            double length = sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2) + pow(p1.z - p2.z, 2));
                            Spring s = {kSpring, p1index, p2index, length, 0, 0, 0};
                            int springIndex = springs.size();
                            springs.push_back(s);
                            int ppsIndex1 = p1index * maxSprings + p1.numSprings;
                            int ppsIndex2 = p2index * maxSprings + p2.numSprings;
                            pointSprings[ppsIndex1] = springIndex;
                            pointSprings[ppsIndex2] = springIndex;
                            points[p1index].numSprings += 1;
                            points[p2index].numSprings += 1;
                        }
                    }
                }
            }
        }
    }
}